
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

__global__ void generate_random_numbers(float *random_numbers, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState state;
    hiprand_init(1234, idx, 0, &state); // init
    if (idx < n) {
        random_numbers[idx] = hiprand_uniform(&state);
    }
}

int main() {
    int n = 1000;
    float *d_random_numbers;
    hipMalloc(&d_random_numbers, n * sizeof(float));
    generate_random_numbers<<<(n + 255) / 256, 256>>>(d_random_numbers, n);
    hipDeviceSynchronize();

    // Copy data back to host and print for verification
    float *h_random_numbers = (float *)malloc(n * sizeof(float));
    hipMemcpy(h_random_numbers, d_random_numbers, n * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < 10; i++) {
        printf("%f\n", h_random_numbers[i]);
    }

    hipFree(d_random_numbers);
    free(h_random_numbers);
    return 0;
}

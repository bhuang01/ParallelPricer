
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

__global__ void generate_random_numbers(float *random_numbers, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState state;
    hiprand_init(1234, idx, 0, &state); // init
    if (idx < n) {
        random_numbers[idx] = hiprand_uniform(&state);
    }
}
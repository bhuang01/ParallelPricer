#include "hip/hip_runtime.h"
#include "random_generator.cuh"
#include "utils.cuh"

__global__ void init_random_states(hiprandState *states, unsigned long long seed, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        hiprand_init(seed, idx, 0, &states[idx]);
    }
}

__global__ void generate_uniform_random_numbers(hiprandState *states, float *random_numbers, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        random_numbers[idx] = hiprand_uniform(&states[idx]);
    }
}

__global__ void generate_normal_random_numbers(hiprandState *states, float *random_numbers, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        random_numbers[idx] = hiprand_normal(&states[idx]);
    }
}

hipError_t generate_random_numbers(float **d_random_numbers, int n, bool use_normal_distribution) {
    hipError_t cudaStatus;

    cudaStatus = hipMalloc(d_random_numbers, n * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return cudaStatus;
    }

    hiprandState *d_states;
    cudaStatus = hipMalloc(&d_states, n * sizeof(hiprandState));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        hipFree(*d_random_numbers);
        return cudaStatus;
    }

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    init_random_states<<<blocksPerGrid, threadsPerBlock>>>(d_states, time(NULL), n);

    if (use_normal_distribution) {
        generate_normal_random_numbers<<<blocksPerGrid, threadsPerBlock>>>(d_states, *d_random_numbers, n);
    } else {
        generate_uniform_random_numbers<<<blocksPerGrid, threadsPerBlock>>>(d_states, *d_random_numbers, n);
    }

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(*d_random_numbers);
        hipFree(d_states);
        return cudaStatus;
    }

    hipFree(d_states);

    return cudaStatus;
}
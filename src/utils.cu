#include "utils.cuh"

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n",
                file, line, static_cast<unsigned int>(result), hipGetErrorString(result), func);
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }
}

void start_timer(hipEvent_t *start, hipEvent_t *stop) {
    CHECK_CUDA_ERROR(hipEventCreate(start));
    CHECK_CUDA_ERROR(hipEventCreate(stop));
    CHECK_CUDA_ERROR(hipEventRecord(*start));
}

float stop_timer(hipEvent_t start, hipEvent_t stop) {
    float milliseconds = 0;
    CHECK_CUDA_ERROR(hipEventRecord(stop));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    CHECK_CUDA_ERROR(hipEventElapsedTime(&milliseconds, start, stop));
    CHECK_CUDA_ERROR(hipEventDestroy(start));
    CHECK_CUDA_ERROR(hipEventDestroy(stop));
    return milliseconds;
}